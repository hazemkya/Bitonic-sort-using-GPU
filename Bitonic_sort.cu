#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <crt/hip/device_functions.h>
#include <time.h>


void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.4fs\n", elapsed);
}

__global__ void sort(int* d_a, int step, int stage, int N, int doPrint) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int seqL = pow(2, step);		// sequnce length per stage
	int n = seqL / (pow(2, stage - 1));
	int shift = n / 2;

	if (idx < N) {
		if (doPrint == 1) printf("thread %d is active.\n", idx);

		if ((idx % n) < shift) { //decide if thread is active or not

			// decide if the sort is ascending or descending 
			/* even | ascending + */
			if ((idx / seqL) % 2 == 0 && idx < N) {
				if (doPrint == 1) printf("(+) ascending | even: has been reached by thread %d, seqL = %d.\n", idx, seqL);
				if (d_a[idx] > d_a[idx + shift]) {
					int temp = d_a[idx];
					d_a[idx] = d_a[idx + shift];
					d_a[idx + shift] = temp;
					if (doPrint == 1) printf("(+) ascending | swaped d_a[%d] = %d , for d_a[%d] = %d\n", idx, d_a[idx], idx + shift, d_a[idx + shift]);
				}
				else {
					if (doPrint == 1) printf("(+) ascending | thread %d did not swap\n", idx);
				}
			}

			/* odd | descending - */
			if ((idx / seqL) % 2 == 1 && idx < N) {
				if (doPrint == 1) printf("(-) descending | odd: has been reached by thread %d, seqL = %d.\n", idx, seqL);
				if (d_a[idx] < d_a[idx + shift]) {
					int temp = d_a[idx];
					d_a[idx] = d_a[idx + shift];
					d_a[idx + shift] = temp;
					if (doPrint == 1) printf("(-) descending | swaped d_a[%d] = %d , for d_a[%d] = %d\n", idx, d_a[idx], idx + shift, d_a[idx + shift]);
				}
				else {
					if (doPrint == 1) printf("(-) descending | thread %d did not swap\n", idx);
				}
			}
		}
		else {
			if (doPrint == 1) printf("thread %d is inactive.\n", idx);
		}
	}
}

void random_ints(int* x, int size)
{
	int i;
	for (i = 0; i < size; i++) {
		x[i] = rand() % 5000;
	}
}

void print_values(int* a, int* b, int N) {
	printf("Before    After\n");
	for (int i = 0; i < N; i++) {
		printf("a[%d]=%d , b[%d]=%d\n", i, a[i], i, b[i]);
	}
}

void bitonic_sort(int N, int blocks, int threads, int* d_a, int doPrint) {

	int itters = log2(N);
	int totalSteps = itters;

	for (int i = 1; i <= totalSteps; i++) {
		if(doPrint == 1) printf("step %d now.\n", i);
		int totalStages = i;		// calculate stages for the current step

		for (int j = 1; j <= totalStages; j++) {
			if (doPrint == 1) printf("stage %d now.\n", j);

			sort << <blocks, threads >> > (d_a, i, j, N, doPrint); // N/2 threads
		}
	}

}

int take_input() {

	int values;
	printf("Please enter the number of valuse:");
	scanf("%d", &values);
	int exp = ceil(log(values) / log(2));
	int N = pow(2, exp);
	int padding = N - values;

	return N;
}

int print_state() {
	int doPrint;
	printf("Do you want to print logs(1= Yes, 0= No)? \n");
	printf("Please note disabling logging will have better performance! \n");
	scanf("%d", &doPrint);
	if (doPrint == 0) return doPrint;
	if (doPrint == 1) return doPrint;
	while (!(doPrint == 0 || doPrint == 1)) {
		printf("Please enter value 1 or 0 (1= Yes, 0= No)\n");
		scanf("%d", &doPrint);
		if (doPrint == 0) return doPrint;
		if (doPrint == 1) return doPrint;
	}
}

void select_device(){
	int select;
	printf("Do you want to select a device(1= Yes, 0= No)? \n");
	printf("If No default is chosen. \n");
	scanf("%d", &select);
	if (select == 1) {
		int nDevices;
		hipGetDeviceCount(&nDevices);
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
		}
		printf("Please enter the device number: \n");
		int chosen = scanf("%d", &chosen);
		hipSetDevice(chosen);
	}
}

int main()
{
	clock_t start, stop;

	int doPrint = print_state();

	select_device();

	int N = take_input();


	int* a, * b;
	int* d_a;
	int size = N * sizeof(int);

	hipMalloc((void**)&d_a, size);

	a = (int*)malloc(size); random_ints(a, N);
	b = (int*)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	int device;
	hipGetDevice(&device);
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device);

	int MaxThreads = props.maxThreadsPerBlock;
	int threads = (int &)MaxThreads;
	int blocks = N / threads + 1;
	printf("Running on: threads=%d , blocks=%d\n", threads,blocks);

	start = clock();
	bitonic_sort(N, blocks, threads, d_a, doPrint);
	stop = clock();

	hipMemcpy(b, d_a, size, hipMemcpyDeviceToHost);

	print_values(a, b, N);

	print_elapsed(start, stop);


	free(a);	free(b);
	hipFree(d_a);

	return 0;
		
	}
